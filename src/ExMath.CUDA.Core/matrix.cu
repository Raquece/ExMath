#include <iostream>
#include <stdexcept>
#include <hip/hip_runtime.h>


/// @brief Kernel method for performing the matrix dot product.
/// @tparam T Type of data in the matrices.
/// @param A Matrix A.
/// @param B Matrix B.
/// @param P Product matrix.
/// @param aRows Number of rows in A.
/// @param aCols Number of columns in A.
/// @param bRows Number of rows in B.
/// @param bCols Number of columns in B.
template<typename T>
__global__ void CUDA_matrixMultiply(T* A, T* B, T* P, int aRows, int aCols, int bRows, int bCols)
{
    int width = aCols; // aCols and bRows are the same

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Grid stride loop to ensure all necessary data is processed, even
    // in abscence of enough threads.
    for (int j = row; j < aRows; j += blockDim.y * gridDim.y)
    {
        if (j < aRows)
        {
            for (int i = col; i < bCols; i += blockDim.x * gridDim.y)
            {
                if (i < bCols)
                {
                    int pIndex = j * bCols + i;

                    T sum = 0;
                    for (int k = 0; k < width; k++)
                    {
                        sum += A[j * aCols + k] * B[k * bCols + i];
                    }
                    
                    P[pIndex] = sum;
                }
                else
                {
                    break;
                }
            }
        }
        else
        {
            break;
        }
    }
}

/// @brief Host code for performing matrix mulitplication on flattened arrays.
/// @tparam T Type of data in the matrices.
/// @param A Matrix A.
/// @param B Matrix B.
/// @param aRows Number of rows in A.
/// @param aCols Number of columns in A.
/// @param bRows Number of rows in B.
/// @param bCols Number of columns in B.
/// @param dim_grid The dim size of the grid.
/// @param dim_block The dim size of the blocks.
/// @return The resultant flattened array.
template<typename T>
T* matrixMultiplyFlattened(T* A, T* B, int aRows, int aCols, int bRows, int bCols, dim3 dim_grid, dim3 dim_block)
{
    int aSize = aRows * aCols;
    int bSize = bRows * aCols;
    int pRows = aRows;
    int pCols = bCols;
    int pSize = pRows * pCols;

    T* P_flattened, *d_A, *d_B, *d_P;

    P_flattened = (T*)malloc(sizeof(T) * pSize);

    hipMalloc(&d_A, sizeof(T) * aSize);
    hipMalloc(&d_B, sizeof(T) * bSize);
    hipMalloc(&d_P, sizeof(T) * pSize);

    hipMemcpy(d_A, A, aSize * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bSize * sizeof(T), hipMemcpyHostToDevice);

    CUDA_matrixMultiply<<<dim_grid, dim_block>>>(d_A, d_B, d_P, aRows, aCols, bRows, bCols);

    hipDeviceSynchronize();

    hipMemcpy(P_flattened, d_P, pSize * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_P);

    return P_flattened;
}

/// @brief Exposes the interface to be exported in the DLL to call different types of matrix multiplication methods depending on the data type of the matrices.
/// @param A Matrix A.
/// @param B Matrix B.
/// @param aRows Number of rows in A.
/// @param aCols Number of columns in A.
/// @param bRows Number of rows in B.
/// @param bCols Number of columns in B.
/// @param dim_grid The dim size of the grid.
/// @param dim_block The dim size of the blocks.
/// @return The resultant flattened array.
void* call_matrixMultiplyFlattened(void* A, void* B, int aRows, int aCols, int bRows, int bCols, dim3 dim_grid, dim3 dim_block, int operationId)
{
    void *o;

    switch (operationId)
    {
    case 1:
        o = matrixMultiplyFlattened((double*)A, (double*)B, aRows, aCols, bRows, bCols, dim_grid, dim_block);
        break;
    case 2:
        o = matrixMultiplyFlattened((int*)A, (int*)B, aRows, aCols, bRows, bCols, dim_grid, dim_block);
        break;
    default:
        break;
    }

    return o;
}