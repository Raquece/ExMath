﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "matrix.cuh"

extern "C"
{
    __declspec(dllexport) void* MatrixMultiply(void* A, void* B, int aRows, int aCols, int bRows, int bCols, int dim_grid_x, int dim_grid_y, int dim_grid_z, int dim_block_x, int dim_block_y, int dim_block_z, int operationId)
    {
        return call_matrixMultiplyFlattened(A, B, aRows, aCols, bRows, bCols, dim3(dim_grid_x, dim_grid_y, dim_grid_z), dim3(dim_block_x, dim_block_y, dim_block_z), operationId);
    }
}